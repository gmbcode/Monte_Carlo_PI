#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <random>
#include <iomanip>
__device__ unsigned long long int countin = 0;
// TODO increase efficiency and utilisation 
__global__ void setup_kernel(int seed,hiprandState *state)
{
  auto id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, id, 0, &state[id]);
}

__global__ void simulate_mc_pi(hiprandState *state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread's local counter
    unsigned long long local_inside = 0;
    
    // Generate random points
    for (unsigned long long i = 0; i < 100000; i++) {
        // Generate random point in unit square [-1, 1] x [-1, 1]
        double x = hiprand_uniform(&state[tid]) * 2.0 - 1.0;
        double y = hiprand_uniform(&state[tid]) * 2.0 - 1.0;
        
        // Check if point is inside unit circle
        if (x * x + y * y <= 1.0) {
            local_inside++;
        }
    }
    
    // Add to global counter atomically
    atomicAdd(&countin, local_inside);
}

__global__ void readerin(unsigned long long int* result){
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = countin;
    }
}

int main() {
    hiprandState* dStates;
    unsigned long long int total_points_sampled = 2048ULL * 256ULL * 100000ULL;
    hipProfilerStart();
    hipMalloc((void **) &dStates, sizeof(hiprandState) * 2048 * 256);  
    std::random_device rd;
    int seed = rd();
    setup_kernel<<<2048,256>>>(seed,dStates);
    hipDeviceSynchronize();
    std :: cout << "Sampling random points now ..." << std :: endl ;
    simulate_mc_pi<<<2048, 256>>>(dStates);
    hipDeviceSynchronize();
    std::cout << "Finished sampling " << total_points_sampled << " points " << std :: endl;
    unsigned long long int* resultin;
    hipMalloc(&resultin, sizeof(unsigned long long int));
    unsigned long long int counter_in;

    readerin<<<1, 1>>>(resultin);
    hipDeviceSynchronize();
    hipMemcpy(&counter_in, resultin, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    std::cout << "Cleaning up..."<< std :: endl;
    hipFree(resultin); 
    hipFree(dStates);
    std::cout << "Finished cleaning up sucessfully"<< std :: endl;
    hipProfilerStop();
    
    std::cout << "Total number of points is : " << total_points_sampled << std :: endl;
    std::cout << "Number of points inside the circle are : " << counter_in << std :: endl;
    long double pi_estimate = 4.0 * (long double)counter_in / (long double)total_points_sampled;
    std::cout << "Estimated Pi : " << std::setprecision (15) << pi_estimate << std::endl;

    return 0;
}