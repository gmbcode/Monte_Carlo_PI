#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
__device__ unsigned long long int countin = 0;
// TODO increase efficiency and utilisation 
__global__ void setup_kernel(hiprandState *state)
{
  auto id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(123456, id, 0, &state[id]);
}

__global__ void simulate_mc_pi(hiprandState *state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread's local counter
    unsigned long long local_inside = 0;
    
    // Generate random points
    for (unsigned long long i = 0; i < 100000; i++) {
        // Generate random point in unit square [-1, 1] x [-1, 1]
        double x = hiprand_uniform(&state[tid]) * 2.0 - 1.0;
        double y = hiprand_uniform(&state[tid]) * 2.0 - 1.0;
        
        // Check if point is inside unit circle
        if (x * x + y * y <= 1.0) {
            local_inside++;
        }
    }
    
    // Add to global counter atomically
    atomicAdd(&countin, local_inside);
}

__global__ void readerin(unsigned long long int* result){
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = countin;
    }
}

int main() {
    hiprandState* dStates;
    hipMalloc((void **) &dStates, sizeof(hiprandState) * 2048 * 256);  
    setup_kernel<<<2048,256>>>(dStates);
    hipDeviceSynchronize();
    simulate_mc_pi<<<2048, 256>>>(dStates);
    hipDeviceSynchronize();
    unsigned long long int* resultin;
    hipMalloc(&resultin, sizeof(unsigned long long int));
    unsigned long long int counter_in;

    readerin<<<1, 1>>>(resultin);
    hipDeviceSynchronize();
    hipMemcpy(&counter_in, resultin, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipFree(resultin); 
    hipFree(dStates);
    unsigned long long int total_points_sampled = 2048ULL * 256ULL * 100000ULL;
    std::cout << "Total number of points is : " << total_points_sampled << std :: endl;
    std::cout << "Number of points inside the circle are : " << counter_in << std :: endl;
    long double pi_estimate = 4.0 * (long double)counter_in / (long double)total_points_sampled;
    std::cout << "Estimated Pi : " << pi_estimate << std::endl;

    return 0;
}